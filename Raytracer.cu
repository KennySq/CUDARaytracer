#include "hip/hip_runtime.h"
#include "Raytracer.cuh"

__device__ DWORD* gPixels;
__device__ HittableList* gWorld;


//void kernelInitConstData(double* constScalar, Vec3* constVector)
//{
//	double aspectRatio = 4.0 / 3.0;
//
//	Point3 origin = Point3(0, 0, 0);
//	Vec3 vertical = Vec3(0, 2.0, 0);
//	Vec3 horizontal = Vec3(vertical.y() * aspectRatio, 0, 0);
//	Vec3 lowerLeft = origin - horizontal / 2 - vertical / 2 - Vec3(0, 0, 1.0);
//
//	double scalars[] = { aspectRatio };
//	Vec3 vectors[] = { origin, vertical, horizontal, lowerLeft };
//
//	hipError_t error = hipMemcpy(constScalar, scalars, sizeof(double) * ARRAYSIZE(scalars), hipMemcpyHostToDevice);
//	cudaErrorCheck(error);
//
//	error = hipMemcpy(constVector, vectors, sizeof(Vec3) * ARRAYSIZE(vectors), hipMemcpyHostToDevice);
//	cudaErrorCheck(error);
//
//}

inline __device__ void setColor(LPDWORD pixels, int width, int height, Color color)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int tid = y * width + x;

	int writeColor = 0;

	int r = color[0] * 255.999;
	int g = color[1] * 255.999;
	int b = color[2] * 255.999;

	int ir = r << 16;
	int ig = g << 8;
	int ib = b;

	writeColor |= ir;
	writeColor |= ig;
	writeColor |= ib;

	pixels[tid] = writeColor;
}


__global__ void kernelClearScreen(LPDWORD pixels, Color color, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int tid = y * width + x;

	int writeColor = 0;

	int r = color[0] * 255.999;
	int g = color[1] * 255.999;
	int b = color[2] * 255.999;

	int ir = r << 16;
	int ig = g << 8;
	int ib = b;

	writeColor |= ir;
	writeColor |= ig;
	writeColor |= ib;

	pixels[tid] = writeColor;
	
}

__global__ void kernelBackground(LPDWORD pixels, int width, int height)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	double u = double(x) / (width);
	double v = double(y) / (height);

	double aspectRatio = 4.0 / 3.0;

	Point3 origin = Point3(0, 0, 0);
	Vec3 vertical = Vec3(0, 2.0, 0);
	Vec3 horizontal = Vec3(vertical.y() * aspectRatio, 0, 0);
	Vec3 lowerLeft = origin - horizontal / 2 - vertical / 2 - Vec3(0, 0, 1.0);

	Ray r(origin, lowerLeft + u * horizontal + v * vertical - origin);

	Color outColor{};
	Vec3 unitDirection = UnitVector(r.mDirection);

	double t = 0.5 * (unitDirection.e[1] + 1.0);
	outColor = (1.0 - t) * Color(1.0, 1.0, 1.0) + t * Color(0.5, 0.7, 1.0);

	setColor(pixels, width, height, outColor);

}

template<typename _Ty>
void CopyDeviceToHost(void* device, void* host, unsigned int count)
{
	hipError_t error = hipMemcpy(host, device, count * sizeof(_Ty), hipMemcpyDeviceToHost);
	cudaErrorCheck(error);
}

__device__ Color RayColor(LPDWORD pixels, const Ray& r, const HittableList& world, Hittable** worldObjects, int width, int height)
{
	HitRecord rec;

//	printf("%p\n", worldObjects);

	if (world.Hit(r, 0, infinity, rec, worldObjects))
	{
		return 0.5 * (rec.normal + Color(1, 1, 1));
	}

	Vec3 UnitDirection = UnitVector(r.mDirection);
	double t = 0.5 * (UnitDirection.y() + 1.0);

	return (1.0 - t) * Color(1, 1, 1) + t * Color(0.5, 0.7, 1.0);

}

__global__ void kernelRender(LPDWORD pixels, int width, int height, const HittableList& world, Hittable** worldObjects)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	double u = double(x) / (width);
	double v = double(y) / (height);

	double aspectRatio = 4.0 / 3.0;

	Point3 origin = Point3(0, 0, 0);
	Vec3 vertical = Vec3(0, 2.0, 0);
	Vec3 horizontal = Vec3(vertical.y() * aspectRatio, 0, 0);
	Vec3 lowerLeft = origin - horizontal / 2 - vertical / 2 - Vec3(0, 0, 1.0);

	Ray r(origin, lowerLeft + u * horizontal + v * vertical - origin);

	printf("%p\n", &world);

	Color out = RayColor(pixels, r, world, worldObjects, width, height);

	setColor(pixels, width, height, out);

	return;
}


Raytracer::Raytracer(HWND handle, HINSTANCE instance, unsigned int width, unsigned int height)
	: mHandle(handle), mInst(instance), mWidth(width), mHeight(height)
{
	BITMAPINFO bitInfo{};

	bitInfo.bmiHeader.biSize = sizeof(BITMAPINFOHEADER);
	bitInfo.bmiHeader.biWidth = width;
	bitInfo.bmiHeader.biHeight = height;
	bitInfo.bmiHeader.biBitCount = 32;
	bitInfo.bmiHeader.biPlanes = 1;
	bitInfo.bmiHeader.biCompression = BI_RGB;

	HDC dc = GetDC(mHandle);

	mBitmap = CreateDIBSection(dc, &bitInfo, DIB_RGB_COLORS, (void**)(&mPixels), nullptr, 0);
	mMemoryDC = CreateCompatibleDC(dc);
	SelectObject(mMemoryDC, mBitmap);
	ReleaseDC(mHandle, dc);

	hipError_t error = hipMalloc((void**)&gPixels, sizeof(DWORD) * width * height);
	cudaErrorCheck(error);

	error = hipMalloc((void**)&gWorld, sizeof(HittableList));
	cudaErrorCheck(error);

	error = hipMalloc((void**)&deviceScene, sizeof(Hittable*) * 1);
	cudaErrorCheck(error);

	error = hipMalloc((void**)&deviceSpheres, sizeof(Sphere) * 1);
	cudaErrorCheck(error);


	//kernelMakeShared << <1, 1 >> > (1, deviceScene, deviceSpheres);


	//AddSphere << <1, 1 >> > (Vec3(0, 0, 0), 5.0, gWorld, deviceScene);
	
	hipDeviceSynchronize();
}


void Raytracer::Run()
{
	dim3 grids = dim3(50, 50, 1);
	dim3 blocks = dim3(16, 12, 1);
	
	//kernelBackground << <grids, blocks >> > (gPixels, mWidth ,mHeight);
	
	kernelRender << <grids, blocks >> > (gPixels, mWidth, mHeight, *gWorld, deviceScene);
	hipDeviceSynchronize();


	
	CopyDeviceToHost<DWORD>(gPixels, mPixels, mWidth * mHeight);

}

void Raytracer::Release()
{
	DeleteDC(mMemoryDC);
	DeleteObject(mBitmap);
}
