#include "hip/hip_runtime.h"
#include"Raytracer.cuh"

#include <windows.h>
#include <tchar.h>
#include<iostream>
#include<string>

#define APP_WIDTH 800
#define APP_HEIGHT 600

LRESULT CALLBACK WndProc(
    _In_ HWND   hWnd,
    _In_ UINT   message,
    _In_ WPARAM wParam,
    _In_ LPARAM lParam
);

HINSTANCE hInst;
HWND hWnd;

Raytracer* rayTracer;

int WINAPI WinMain(HINSTANCE hInstance,
    HINSTANCE hPrevInstance,
    LPSTR lpCmdLine,
    int nCmdShow)
{
    static TCHAR szWindowClass[] = _T("DesktopApp");
    static TCHAR szTitle[] = _T("CUDA Raytracer");

    WNDCLASSEX wcex;

    wcex.cbSize = sizeof(WNDCLASSEX);
    wcex.style = CS_HREDRAW | CS_VREDRAW;
    wcex.lpfnWndProc = WndProc;
    wcex.cbClsExtra = 0;
    wcex.cbWndExtra = 0;
    wcex.hInstance = hInstance;
    wcex.hIcon = LoadIcon(wcex.hInstance, IDI_APPLICATION);
    wcex.hCursor = LoadCursor(NULL, IDC_ARROW);
    wcex.hbrBackground = (HBRUSH)(COLOR_WINDOW + 1);
    wcex.lpszMenuName = NULL;
    wcex.lpszClassName = szWindowClass;
    wcex.hIconSm = LoadIcon(wcex.hInstance, IDI_APPLICATION);

    if (!RegisterClassEx(&wcex))
    {
        MessageBox(NULL,
            _T("Call to RegisterClassEx failed!"),
            _T("Windows Desktop Guided Tour"),
            NULL);

        return 1;
    }

    // Store instance handle in our global variable
    hInst = hInstance;

    // The parameters to CreateWindowEx explained:
    // WS_EX_OVERLAPPEDWINDOW : An optional extended window style.
    // szWindowClass: the name of the application
    // szTitle: the text that appears in the title bar
    // WS_OVERLAPPEDWINDOW: the type of window to create
    // CW_USEDEFAULT, CW_USEDEFAULT: initial position (x, y)
    // 500, 100: initial size (width, length)
    // NULL: the parent of this window
    // NULL: this application dows not have a menu bar
    // hInstance: the first parameter from WinMain
    // NULL: not used in this application
    HWND hWnd = CreateWindowEx(
        WS_EX_OVERLAPPEDWINDOW,
        szWindowClass,
        szTitle,
        WS_OVERLAPPEDWINDOW,
        CW_USEDEFAULT, CW_USEDEFAULT,
        APP_WIDTH, APP_HEIGHT,
        NULL,
        NULL,
        hInstance,
        NULL
    );

    if (!hWnd)
    {
        MessageBox(NULL,
            _T("Call to CreateWindow failed!"),
            _T("Windows Desktop Guided Tour"),
            NULL);

        return 1;
    }

    rayTracer = new Raytracer(hWnd, hInst, APP_WIDTH, APP_HEIGHT);

    // The parameters to ShowWindow explained:
    // hWnd: the value returned from CreateWindow
    // nCmdShow: the fourth parameter from WinMain
    ShowWindow(hWnd,
        nCmdShow);

    AllocConsole();

    UpdateWindow(hWnd);

    // Main message loop:
    MSG msg;
    while (GetMessage(&msg, NULL, 0, 0))
    {
        TranslateMessage(&msg);
        DispatchMessage(&msg);
    }

    return (int)msg.wParam;
}

clock_t start, end, duration;

LRESULT WndProc(HWND hWnd, UINT message, WPARAM wParam, LPARAM lParam)
{
    PAINTSTRUCT ps;
    HDC hdc;
    std::string str;
    switch (message)
    {
    case WM_CREATE:
    {
        SetTimer(hWnd, 1, 100, nullptr);
    }
    break;
    case WM_TIMER:
    {
        start = clock();

        rayTracer->Run();
        end = clock();

        duration = (double)end - start;

        InvalidateRgn(hWnd, nullptr, true);
    }
    break;
    case WM_PAINT:
        hdc = BeginPaint(hWnd, &ps);


        BitBlt(hdc, 0, 0, APP_WIDTH, APP_HEIGHT, rayTracer->GetMemoryDC(), 0, 0, SRCCOPY);
        
        str = std::to_string((double)(duration));

        str += " ms";

        TextOut(hdc, 0, 0, str.c_str(), str.size());


        EndPaint(hWnd, &ps);
        break;
    case WM_DESTROY:
        rayTracer->Release();
        PostQuitMessage(0);
        break;
    default:
        return DefWindowProc(hWnd, message, wParam, lParam);
        break;
    }

    return 0;
}
